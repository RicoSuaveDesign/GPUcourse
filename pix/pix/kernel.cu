#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "highperformancetimer.h"

#include <stdio.h>

using namespace std;
using namespace cv;

typedef unsigned char ubyte;

void threshold(unsigned char threshold, int width, int height, unsigned char* data);
void threshold(unsigned char threshold, Mat &image);
hipError_t thresholdGPU(unsigned char threshold, Mat &image);
void BoxFilter(ubyte *s, ubyte *d, int w, int h, int* k, int kw, int kh, ubyte *temp);


// shitty goddamned bad global variables
HighPrecisionTime hpt;
unsigned char* dev_image = nullptr;
unsigned char* dev_moddedimage = nullptr; 
Mat image;
int Threshold_slider = 128;
int Box_Slider = 1;
float totalTime = 0.0; // remember to reset this every time a new timer is called
int timesCalled = 0;
int ke[9] = { -1,0,1,-2,0,2,-1,0,1 };
int k2[9] = { -1, 2,-1, 0,0,0, 1,2,1 };
int boxk[25];
int kh = 5;
int kw = 5;


__global__ void threshKernel(unsigned char * image, unsigned char* moddedimage, int size, int threshold)
{
	// multiply by blockdimx because it just werks i guess 
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		if (image[i] > threshold)
		{
			moddedimage[i] = 255;
		}
		else
		{
			moddedimage[i] = 0;
		}
	}
}

void on_trackbar(int, void*)
{
	//hipError_t cudaStatus;
	//int blocks_needed = (1023 + image.rows * image.cols) / 1024;
	//// call the kernel on the now global device variables
	//threshKernel <<<blocks_needed, 1024 >>> (dev_image, dev_moddedimage, (image.rows * image.cols), Threshold_slider);
	//cudaStatus = hipDeviceSynchronize();
	//
	//cudaStatus = hipMemcpy(image.data, dev_moddedimage, (image.rows * image.cols), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess)
	//{
	//	cerr << "Memcpy from GPU to CPU failed!" << endl;
	//	hipFree(dev_image);
	//	hipFree(dev_moddedimage);
	//}
	//cout << Threshold_slider << endl;
	//BoxFilter(s, d, image.cols, image.rows, k, 3, 3, temp);
	cout << hpt.TimeSinceLastCall();
	imshow("Display window", image);

}
void box_trackbar(int, void*);


int main(int argc, char** argv)
{
	if (argc != 2)
	{
		cout << "Usage: display_image ImageToLoadAndDisplay" << endl;
		return -1;
	}
	for (int i = 0; i < 25; i++)
	{
		boxk[i] = 1;
	}
	//set up cuda stuff so it only needs to happen once 
	hipDeviceProp_t devProp;
	hipSetDevice(0);
	hipGetDeviceProperties(&devProp, 0);

	//Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_COLOR); // read the file

	cout << "Number of channels: " << image.channels() << endl;
	if (!image.data)
	{
		cout << "Could not open or find the image" << endl;
		return -1;
	}

	
	cvtColor(image, image, cv::COLOR_RGB2GRAY);
	ubyte *src = image.data;
	ubyte *dst = image.data;
	ubyte *temp = image.data;
	
	namedWindow("Display window", WINDOW_NORMAL);
	imshow("Display window", image);
	waitKey(0);
	hpt.TimeSinceLastCall();
	BoxFilter(src, dst, image.cols, image.rows,  boxk, kh, kw, temp);
	cout << "The box filter took " << hpt.TimeSinceLastCall() << " seconds." << endl;
	//threshold(Threshold, image);
	//hipError_t cudaStatus;
	//cudaStatus = thresholdGPU(Threshold_slider, image);
	//if (cudaStatus != hipSuccess)
	//	cout << "Failed to apply threshold filter" << endl;
	//else
	//{

	//	namedWindow("Display window", WINDOW_NORMAL); //create window for display
	//	imshow("Display window", image); // show image inside it
	//}
	createTrackbar("Threshold", "Display window", &Box_Slider, 10, box_trackbar);
	imshow("Display window", image);
	box_trackbar(Box_Slider, 0);

	waitKey(0); // wait for keystroke in window

	cout << endl << "Final average: " << totalTime / timesCalled << " seconds" << endl;
	cout << "image size: " << image.cols << " x " << image.rows << endl;
	cout << "kernel size: 3 x 3" << endl;

#ifdef _WIN32 || _WIN64
	system("pause");
#endif

	//hipFree(dev_image); // and here we are freein the memory on gpu
	//hipFree(dev_moddedimage);
	return 0;


}

void box_trackbar(int, void*)
{
	int *p_k = boxk;

	ubyte *s = image.data;
	Mat d;
	image.copyTo(d);
	ubyte *tempo = image.data;
	

	hpt.TimeSinceLastCall();
	BoxFilter(s, d.data, image.cols, image.rows, p_k, kh, kw, tempo);
	float currentTime = hpt.TimeSinceLastCall();
	totalTime += currentTime;
	timesCalled++;

	cout << "Time this run: " << currentTime << " seconds" << endl;
	cout << "Current average: " << totalTime / timesCalled << endl;

	imshow("Display window", image);
	
}

void BoxFilter(ubyte *s, ubyte *d, int w, int h, int *k, int kw, int kh, ubyte *temp)
{
	
	// later on we divide by the sum of all the values in the box kernel -- so calculate it now
	int kernelSum = 0;
	for (int i = 0; i < kw*kh; i++)
	{
		kernelSum += k[i];
	}


	// this makes calculating relative indices (e.g. what is one value of "up" to a 1D array?) a one time task, or at the very least a much more readable operation
	//int indices[9] = { -(w + 1),  -w, -(w - 1), -1, 0,  +1,	w - 1, w,  w + 1 };
	
	// calculates our image edges -- wedge is width edge, hedge is height edge

	int kwedge = kw / 2;
	int khedge = kh / 2;

	int indexOffset;
	for (int i = khedge; i < h - khedge; i++)
	{
		for (int j = kwedge; j < w - kwedge; j++)
		{
			// first we start with current, which starts at 0.0. Then we calculate the relative ups, downs, etc with indexoffset.
			float current = 0.0f;
			indexOffset = (i*w) + j;
			for (int ki = -khedge; ki <= khedge; ki++)
			{
				for (int kj = -kwedge; kj <= kwedge; kj++)
				{
					// relative pixel is found by multiplying the current vertical kernel pixel by image width, and then adding the current kernel horizontal index
					int relativepixel = ki * w + kj;
					// kernel pixel is current kernel height plus vertical edge, then multiplied  by kernel hiehgt, which then current kernel width is added to horizontal edge
					int kernelpix = (ki + khedge) * kw + kj + kwedge;
					// current gets the value of the current pixel and multiplies by the value in the current index of the kernel
					current += float(s[indexOffset + relativepixel]) * float(k[kernelpix]);
				}
			}
			if (kernelSum != 0)
			{
				// output image pixels all are divided by kernel sum which is 9
				d[indexOffset] = int(current / (float)kernelSum);
			}
			else
			{
				d[indexOffset] = int(current / 1.0f);
			}
		}
	}
}

void threshold(unsigned char threshold, int width, int height, unsigned char* data)
{
	unsigned char* end_data = (data + (width * height) + width);
	for (unsigned char* p = data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

void threshold(unsigned char threshold, Mat &image)
{
	unsigned char* end_data = (image.data + (image.cols * image.rows) + image.cols);
	for (unsigned char* p = image.data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

hipError_t thresholdGPU(unsigned char threshold, Mat &image)
{

	hipError_t cudaStatus;
	int size = image.rows * image.cols *sizeof(unsigned char);

	// declare and then allocate GPU memory
	//unsigned char* dev_image = nullptr;
	//unsigned char* dev_moddedimage = nullptr;
	try
	{
		cudaStatus = hipMalloc((void**)&dev_image, (size));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_image!";
		}
		cudaStatus = hipMalloc((void**)&dev_moddedimage, (size));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_moddedimage!";
		}
	}
	catch (char* message)
	{
		cerr << message << endl;
		if (dev_image != nullptr)
			hipFree(dev_image);
		if (dev_moddedimage != nullptr)
			hipFree(dev_moddedimage);
		cerr << "Freed Cuda Memory" << endl;
	}

	//copy orig image to GPU
	cudaStatus = hipMemcpy(dev_image, image.data, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cerr << "Memcpy from CPU to GPU failed!" << endl;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
	}


	int blocks_needed = (image.rows * image.cols + 1023) / 1024;
	cout << "There will be " << blocks_needed << " blocks with 1024 threads each." << endl;

	threshKernel << <blocks_needed, 1024 >> > (dev_image, dev_moddedimage, size, threshold);
	try
	{
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw "addKernel launch failed!";

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			throw "cudaDeviceSync Failed!";
		}
		cudaStatus = hipMemcpy((unsigned char*)image.data, dev_moddedimage, size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw "hipMemcpy failed!";
		}
	}
	catch (char* err_mess)
	{
		// just cout the error message for now cause we gon free the memory anyway
		cerr << err_mess;
	}


	return cudaStatus;
}


