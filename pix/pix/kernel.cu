#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "highperformancetimer.h"
#include <stdio.h>

using namespace std;
using namespace cv;

typedef unsigned char ubyte;

void threshold(unsigned char threshold, int width, int height, unsigned char* data);
void threshold(unsigned char threshold, Mat &image);
hipError_t thresholdGPU(unsigned char threshold, Mat &image);
void BoxFilter(ubyte *s, ubyte *d, int w, int h, int* k, int kw, int kh, ubyte *temp);
void box_trackbar(int, void*);
void BoxGPU(ubyte *s, ubyte *d, int w, int h, int* k, int kw, int kh, ubyte *temp);


// global variables, which are generally Bad (tm) but we are going to use anyway
HighPrecisionTime hpt;
hipDeviceProp_t devProp;
unsigned char* dev_image = nullptr;
unsigned char* dev_moddedimage = nullptr; 
Mat image;
int Threshold_slider = 128;
int Box_Slider = 1;
float totalTime = 0.0; // remember to reset this every time a new timer is called
int timesCalled = 0;
int ke[9] = { -1,0,1,-2,0,2,-1,0,1 };
int k2[9] = { -1, 2,-1, 0,0,0, 1,2,1 };
// only use positive odd numbers for kh and kw or else it will break
const int kh = 21;
const int kw = 21;
int boxk[kh*kw];


// GPU constants
__constant__ int kernel[kw*kh];
__constant__ float kernelsum;


__global__ void boxKernelGPU(unsigned char * src, unsigned char* dst, int h, int w, int kh, int kw)
{
	//this is a two dimensional problem so we gon use blockx and blocky
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	//just like in the CPU version, we calculate the distance from the vertical and horizontal edges of the kernel
	int khedge = kh / 2;
	int kwedge = kw / 2;
	int indexOffset = (j*w) + i;

	if (indexOffset < (w*h))
	{

		float current = 0.0f;

		for (int ki = -khedge; ki <= khedge; ki++)
		{
			for (int kj = -kwedge; kj <= kwedge; kj++)
			{
				// relative pixel is found by multiplying the current kernel row by image width, and then adding the current kernel column
				int relativepixel = ki * w + kj;
				// kernel pixel is current kernel height plus vertical edge, then multiplied  by kernel height, which then current kernel width is added to horizontal edge
				int kernelpix = (ki + khedge) * kw + kj + kwedge;
				// current gets the value of the current pixel and multiplies by the value in the current index of the kernel
				current += float(src[indexOffset + relativepixel]) * float(kernel[kernelpix]);
			}
		}
		if (kernelsum != 0)
		{
			// output image pixels all are divided by kernel sum which is 9 in a 3x3 box filter
			dst[indexOffset] = int(current / (float)kernelsum);
		}
		else
		{
			dst[indexOffset] = int(current / 1.0f);
		}
	}

}

__global__ void threshKernel(unsigned char * image, unsigned char* moddedimage, int size, int threshold)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		if (image[i] > threshold)
		{
			moddedimage[i] = 255;
		}
		else
		{
			moddedimage[i] = 0;
		}
	}
}

void on_trackbar(int, void*)
{
	//hipError_t cudaStatus;
	//int blocks_needed = (1023 + image.rows * image.cols) / 1024;
	//// call the kernel on the now global device variables
	//threshKernel <<<blocks_needed, 1024 >>> (dev_image, dev_moddedimage, (image.rows * image.cols), Threshold_slider);
	//cudaStatus = hipDeviceSynchronize();
	//
	//cudaStatus = hipMemcpy(image.data, dev_moddedimage, (image.rows * image.cols), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess)
	//{
	//	cerr << "Memcpy from GPU to CPU failed!" << endl;
	//	hipFree(dev_image);
	//	hipFree(dev_moddedimage);
	//}
	//cout << Threshold_slider << endl;
	//BoxFilter(s, d, image.cols, image.rows, k, 3, 3, temp);
	cout << hpt.TimeSinceLastCall();
	imshow("Display window", image);

}


int main(int argc, char** argv)
{
	if (argc != 2)
	{
		cout << "Usage: display_image ImageToLoadAndDisplay" << endl;
		return -1;
	}
	for (int i = 0; i < kw*kh; i++)
	{
		boxk[i] = 1;
	}

	//set up cuda stuff so it only needs to happen once 
	hipError_t cudaStatus;
	hipSetDevice(0);
	cudaStatus = hipGetDeviceProperties(&devProp, 0);
	if (cudaStatus != hipSuccess)
	{
		cerr << "Graphics card not detected. Are you using a Cuda capable graphics card?" << endl;
		exit(1);
	}


	// image is now global so just imread it now
	image = imread(argv[1], CV_LOAD_IMAGE_COLOR); // read the file
	cout << "Number of channels: " << image.channels() << endl;
	if (!image.data)
	{
		cout << "Could not open or find the image" << endl;
		return -1;
	}

	
	cvtColor(image, image, cv::COLOR_RGB2GRAY);
	ubyte *src = image.data;
	ubyte *dst = image.data;
	ubyte *temp = image.data;
	
	namedWindow("Display window", WINDOW_NORMAL);
	imshow("Display window", image);
	waitKey(0);
	hpt.TimeSinceLastCall();
	BoxGPU(src, dst, image.cols, image.rows,  boxk, kw, kh, temp);
	cout << "The box filter took " << hpt.TimeSinceLastCall() << " seconds." << endl;

	//threshold(Threshold, image);
	//
	//cudaStatus = thresholdGPU(Threshold_slider, image);
	//if (cudaStatus != hipSuccess)
	//	cout << "Failed to apply threshold filter" << endl;
	//else
	//{

	//	namedWindow("Display window", WINDOW_NORMAL); //create window for display
	//	imshow("Display window", image); // show image inside it
	//}

	createTrackbar("Box", "Display window", &Box_Slider, 10, box_trackbar);
	imshow("Display window", image);
	box_trackbar(Box_Slider, 0);

	waitKey(0); // wait for keystroke in window

	cout << endl << "Final average: " << totalTime / timesCalled << " seconds" << endl;
	cout << "image size: " << image.cols << " x " << image.rows << endl;
	cout << "kernel size: 3 x 3" << endl;

#ifdef _WIN32 || _WIN64
	system("pause");
#endif

	return 0;


}

void threshold(unsigned char threshold, int width, int height, unsigned char* data)
{
	unsigned char* end_data = (data + (width * height) + width);
	for (unsigned char* p = data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

void threshold(unsigned char threshold, Mat &image)
{
	unsigned char* end_data = (image.data + (image.cols * image.rows) + image.cols);
	for (unsigned char* p = image.data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

hipError_t thresholdGPU(unsigned char threshold, Mat &image)
{

	hipError_t cudaStatus;
	int maxthreads = devProp.maxThreadsPerBlock;
	int size = image.rows * image.cols * sizeof(unsigned char);
// since the GPU variables are declared globally, we're just going to malloc them here.
	try
	{
		cudaStatus = hipMalloc((void**)&dev_image, (size));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_image!";
		}
		cudaStatus = hipMalloc((void**)&dev_moddedimage, (size));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_moddedimage!";
		}
	}
	catch (char* message)
	{
		cerr << message << endl;
		if (dev_image != nullptr)
			hipFree(dev_image);
		if (dev_moddedimage != nullptr)
			hipFree(dev_moddedimage);
		hipDeviceReset();
		cerr << "Freed Cuda Memory" << endl;
	}

	//copy orig image to GPU
	cudaStatus = hipMemcpy(dev_image, image.data, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cerr << "Memcpy from CPU to GPU failed!" << endl;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
		hipDeviceReset();
		exit(1);
	}


	int blocks_needed = (image.rows * image.cols + (maxthreads - 1)) / maxthreads;
	cout << "There will be " << blocks_needed << " blocks with " << maxthreads << " threads each." << endl;

	threshKernel << <blocks_needed, maxthreads >> > (dev_image, dev_moddedimage, size, threshold);
	try
	{
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw "threshKernel launch failed!";

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			throw "cudaDeviceSync Failed!";
		}
		cudaStatus = hipMemcpy((unsigned char*)image.data, dev_moddedimage, size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw "hipMemcpy failed!";
		}
	}
	catch (char* err_mess)
	{
		cerr << err_mess;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
		hipDeviceReset();
		exit(1);
	}

	hipFree(dev_image);
	hipFree(dev_moddedimage);
	hipDeviceReset();
	return cudaStatus;
}

void BoxFilter(ubyte *s, ubyte *d, int w, int h, int *k, int kw, int kh, ubyte *temp)
{

	// later on we divide by the sum of all the values in the box kernel -- so calculate it now
	int kernelSum = 0;
	for (int i = 0; i < kw*kh; i++)
	{
		kernelSum += k[i];
	}

	// calculates our image edges -- wedge is width edge, hedge is height edge
	int kwedge = kw / 2;
	int khedge = kh / 2;
	int indexOffset;

	for (int i = khedge; i < h - khedge; i++)
	{
		for (int j = kwedge; j < w - kwedge; j++)
		{
			// first we start with current, which starts at 0.0. Then we calculate the relative ups, downs, etc with indexoffset.
			float current = 0.0f;
			indexOffset = (i*w) + j;
			for (int ki = -khedge; ki <= khedge; ki++)
			{
				for (int kj = -kwedge; kj <= kwedge; kj++)
				{
					// relative pixel is found by multiplying the current kernel row by image width, and then adding the current kernel column
					int relativepixel = ki * w + kj;
					// kernel pixel is current kernel height plus vertical edge, then multiplied  by kernel height, which then current kernel width is added to horizontal edge
					int kernelpix = (ki + khedge) * kw + kj + kwedge;
					// current gets the value of the current pixel and multiplies by the value in the current index of the kernel
					current += float(s[indexOffset + relativepixel]) * float(k[kernelpix]);
				}
			}
			if (kernelSum != 0)
			{
				// output image pixels all are divided by kernel sum which is 9 in a 3x3 box filter
				d[indexOffset] = int(current / (float)kernelSum);
			}
			else
			{
				d[indexOffset] = int(current / 1.0f);
			}
		}
	}
}

void box_trackbar(int, void*)
{
	// this is so that we don't continually add blur until it darkens into an unrecognizable image.
	ubyte *s = image.data;
	Mat d;
	image.copyTo(d);
	ubyte *tempo = image.data;


	hpt.TimeSinceLastCall();
	BoxGPU(s, d.data, image.cols, image.rows, boxk, kh, kw, tempo);
	float currentTime = hpt.TimeSinceLastCall();
	totalTime += currentTime;
	timesCalled++;

	cout << "Time this run: " << currentTime << " seconds" << endl;
	cout << "Current average: " << totalTime / timesCalled << endl;

	imshow("Display window", image);

}

void BoxGPU(ubyte *s, ubyte *d, int w, int h, int* k, int kw, int kh, ubyte *temp)
{

	hipError_t cudaStatus;
	int size = image.rows * image.cols * sizeof(unsigned char);
	float hostkernelSum = 0;
	int maxthreads = devProp.maxThreadsPerBlock;

	// cuda device stuff is in main so thats gucci. first we gotta cudamalloc. 
	try
	{
		cudaStatus = hipMalloc((void**)&dev_image, (size));
		if (cudaStatus != hipSuccess)
		{
			throw("hipMalloc failed on dev_image!");
		}
		cudaStatus = hipMalloc((void**)&dev_moddedimage, (size));
		if (cudaStatus != hipSuccess)
		{
			throw("hipMalloc failed on dev_moddedimage!");
		}

		// then we cuda memcpy, src to dev src. modded image just sits pretty and open until we output the changes done by box filter. Or whatever convolving filter we choose when filling the array.
		cudaStatus = hipMemcpy(dev_image, s, size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			throw("hipMemcpy failed from host to dev_image!");
		}

		// we also gotta copy two constants: the kernel ptr and then the kernel sum. The dev constants are declared up in global vars
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(*kernel), k, kw*kh * sizeof(int), 0, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			throw("cudaMemcpytoSymbol failed on the kernel!");
		}

		// if we make it this far, it's now worth it to do a for loop
		for (int i = 0; i < kw*kh; i++)
		{
			hostkernelSum += (float)k[i];
		}

		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(kernelsum), &hostkernelSum, sizeof(float), 0, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			throw("cudaMemcpytoSymbol failed on the kernel sum!");
		}
	}
	catch (char* err_mess)
	{
		cerr << err_mess;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
		hipDeviceReset();
		exit(1);
		
	}

	// determine the blocks needed as well. 
	int blocks_needed = (image.rows * image.cols + (maxthreads - 1)) / maxthreads;
	cout << "There will be " << blocks_needed << " blocks with " << maxthreads << " threads each." << endl;

	// Then we do the kernel. The outer for loops are taken care of by two dimensional block stuff. the inner loops will be literally the same as the cpu implementation.
	boxKernelGPU<<<blocks_needed, 1024>>>(dev_image, dev_moddedimage, image.rows, image.cols, kh, kw);

	try
	{
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw "boxKernelGPU launch failed!";

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			throw "cudaDeviceSync Failed!";
		}
		// cudamemcopy back to host.
		cudaStatus = hipMemcpy((unsigned char*)image.data, dev_moddedimage, size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw "hipMemcpy failed!";
		}
	}
	catch (char* err_mess)
	{
		cerr << err_mess;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
		hipDeviceReset();
		exit(1);
	}

	hipFree(dev_image);
	hipFree(dev_moddedimage);
	hipDeviceReset();
}








