#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

using namespace std;
using namespace cv;

void threshold(unsigned char threshold, int width, int height, unsigned char* data);
void threshold(unsigned char threshold, Mat &image);
hipError_t thresholdGPU(unsigned char threshold, Mat &image);


// shitty goddamned bad global variables
unsigned char* dev_image = nullptr;
unsigned char* dev_moddedimage = nullptr; 
Mat image;
int Threshold_slider = 128;

__global__ void threshKernel(unsigned char * image, unsigned char* moddedimage, int size, int threshold)
{
	// multiply by blockdimx because it just werks i guess 
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		if (image[i] > threshold)
		{
			moddedimage[i] = 255;
		}
		else
		{
			moddedimage[i] = 0;
		}
	}
}

void on_trackbar(int, void*)
{
	hipError_t cudaStatus;
	int blocks_needed = (1023 + image.rows * image.cols) / 1024;
	// call the kernel on the now global device variables
	threshKernel <<<blocks_needed, 1024 >>> (dev_image, dev_moddedimage, (image.rows * image.cols), Threshold_slider);
	cudaStatus = hipDeviceSynchronize();
	
	cudaStatus = hipMemcpy(image.data, dev_moddedimage, (image.rows * image.cols), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		cerr << "Memcpy from GPU to CPU failed!" << endl;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
	}
	cout << Threshold_slider << endl;
	imshow("Display window", image);

}


int main(int argc, char** argv)
{
	if (argc != 2)
	{
		cout << "Usage: display_image ImageToLoadAndDisplay" << endl;
		return -1;
	}
	//set up cuda stuff so it only needs to happen once 
	hipDeviceProp_t devProp;
	hipSetDevice(0);
	hipGetDeviceProperties(&devProp, 0);

	//Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_COLOR); // read the file

	cout << "Number of channels: " << image.channels() << endl;
	if (!image.data)
	{
		cout << "Could not open or find the image" << endl;
		return -1;
	}

	cvtColor(image, image, cv::COLOR_RGB2GRAY);

	//threshold(Threshold, image);
	hipError_t cudaStatus;
	cudaStatus = thresholdGPU(Threshold_slider, image);
	if (cudaStatus != hipSuccess)
		cout << "Failed to apply threshold filter" << endl;
	//else
	//{

	//	namedWindow("Display window", WINDOW_NORMAL); //create window for display
	//	imshow("Display window", image); // show image inside it
	//}
	namedWindow("Display window", WINDOW_NORMAL); //create window for display
	createTrackbar("Threshold", "Display window", &Threshold_slider, 255, on_trackbar);
	imshow("Display window", image);
	on_trackbar(Threshold_slider, 0);

	waitKey(0); // wait for keystroke in window

	hipFree(dev_image); // and here we are freein the memory on gpu
	hipFree(dev_moddedimage);
	return 0;


}

void threshold(unsigned char threshold, int width, int height, unsigned char* data)
{
	unsigned char* end_data = (data + (width * height) + width);
	for (unsigned char* p = data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

void threshold(unsigned char threshold, Mat &image)
{
	unsigned char* end_data = (image.data + (image.cols * image.rows) + image.cols);
	for (unsigned char* p = image.data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

hipError_t thresholdGPU(unsigned char threshold, Mat &image)
{

	hipError_t cudaStatus;
	int size = image.rows * image.cols *sizeof(unsigned char);

	// declare and then allocate GPU memory
	//unsigned char* dev_image = nullptr;
	//unsigned char* dev_moddedimage = nullptr;
	try
	{
		cudaStatus = hipMalloc((void**)&dev_image, (size));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_image!";
		}
		cudaStatus = hipMalloc((void**)&dev_moddedimage, (size));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_moddedimage!";
		}
	}
	catch (char* message)
	{
		cerr << message << endl;
		if (dev_image != nullptr)
			hipFree(dev_image);
		if (dev_moddedimage != nullptr)
			hipFree(dev_moddedimage);
		cerr << "Freed Cuda Memory" << endl;
	}

	//copy orig image to GPU
	cudaStatus = hipMemcpy(dev_image, image.data, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cerr << "Memcpy from CPU to GPU failed!" << endl;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
	}


	int blocks_needed = (image.rows * image.cols + 1023) / 1024;
	cout << "There will be " << blocks_needed << " blocks with 1024 threads each." << endl;

	threshKernel << <blocks_needed, 1024 >> > (dev_image, dev_moddedimage, size, threshold);
	try
	{
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw "addKernel launch failed!";

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			throw "cudaDeviceSync Failed!";
		}
		cudaStatus = hipMemcpy((unsigned char*)image.data, dev_moddedimage, size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw "hipMemcpy failed!";
		}
	}
	catch (char* err_mess)
	{
		// just cout the error message for now cause we gon free the memory anyway
		cerr << err_mess;
	}


	return cudaStatus;
}


