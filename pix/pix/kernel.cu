#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

using namespace std;
using namespace cv;

void threshold(unsigned char threshold, int width, int height, unsigned char* data);
void threshold(unsigned char threshold, Mat &image);
hipError_t thresholdGPU(unsigned char threshold, Mat &image);

__global__ void threshKernel(unsigned char * image, unsigned char* moddedimage, int size, unsigned char threshold)
{
	int i = blockDim.x * gridDim.x + threadIdx.x;
	if (i < size)
	{
		if (image[i] > threshold)
		{
			moddedimage[i] = 255;
		}
		else
		{
			moddedimage[i] = 0;
		}
	}
}


int main(int argc, char** argv)
{
	if (argc != 2)
	{
		cout << "Usage: display_image ImageToLoadAndDisplay" << endl;
		return -1;
	}
	//set up cuda stuff so it only needs to happen once 
	hipDeviceProp_t devProp;
	hipSetDevice(0);
	hipGetDeviceProperties(&devProp, 0);

	Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_COLOR); // read the file

	cout << "Number of channels: " << image.channels() << endl;
	if (!image.data)
	{
		cout << "Could not open or find the image" << endl;
		return -1;
	}

	cvtColor(image, image, cv::COLOR_RGB2GRAY);
	unsigned char Threshold = 128;
	//threshold(Threshold, image);
	hipError_t cudaStatus;
	cudaStatus = thresholdGPU(Threshold, image);
	if (cudaStatus != hipSuccess)
		cout << "Failed to apply threshold filter" << endl;
	//else
	//{

	//	namedWindow("Display window", WINDOW_NORMAL); //create window for display
	//	imshow("Display window", image); // show image inside it
	//}
	waitKey(0); // wait for keystroke in window
	return 0;


}

void threshold(unsigned char threshold, int width, int height, unsigned char* data)
{
	unsigned char* end_data = (data + (width * height) + width);
	for (unsigned char* p = data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

void threshold(unsigned char threshold, Mat &image)
{
	unsigned char* end_data = (image.data + (image.cols * image.rows) + image.cols);
	for (unsigned char* p = image.data; p < end_data; p++)
	{
		if (*p > threshold)
		{
			*p = 255;
		}
		else
		{
			*p = 0;
		}
	}
}

hipError_t thresholdGPU(unsigned char threshold, Mat &image)
{

	hipError_t cudaStatus;
	int size = image.rows * image.cols *sizeof(unsigned char);

	// allocates on cpu as an unsigned char
	unsigned char* imagedata = image.data;

	// declare and then allocate GPU memory
	unsigned char* dev_image = nullptr;
	unsigned char* dev_moddedimage = nullptr;
	try
	{
		cudaStatus = hipMalloc((void**)&dev_image, (size * sizeof(unsigned char)));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_image!";
		}
		cudaStatus = hipMalloc((void**)&dev_moddedimage, (size * sizeof(unsigned char)));
		if (cudaStatus != hipSuccess)
		{
			throw "hipMalloc failed on dev_moddedimage!";
		}
	}
	catch (char* message)
	{
		cerr << message << endl;
		if (dev_image != nullptr)
			hipFree(dev_image);
		if (dev_moddedimage != nullptr)
			hipFree(dev_moddedimage);
		cerr << "Freed Cuda Memory" << endl;
	}

	//copy orig image to GPU
	cudaStatus = hipMemcpy(dev_image, imagedata, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cerr << "Memcpy from CPU to GPU failed!" << endl;
		hipFree(dev_image);
		hipFree(dev_moddedimage);
	}

	int blocks_needed = size / 1024;
	cout << "There will be " << blocks_needed << " blocks with 1024 threads each." << endl;

	threshKernel << <blocks_needed, 1024 >> > (dev_image, dev_moddedimage, size, threshold);

	try
	{
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			throw "addKernel launch failed!";

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			throw "cudaDeviceSync Failed!";
		}
		cudaStatus = hipMemcpy((unsigned char*)image.data, dev_moddedimage, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw "hipMemcpy failed!";
		}
	}
	catch (char* err_mess)
	{
		// just cout the error message for now cause we gon free the memory anyway
		cerr << err_mess;
	}
	hipFree(dev_image);
	hipFree(dev_moddedimage);

	if (cudaStatus == hipSuccess)
	{
		namedWindow("Display window", WINDOW_NORMAL); //create window for display
		imshow("Display window", image); // show image inside it
	}

	return cudaStatus;





	
}


